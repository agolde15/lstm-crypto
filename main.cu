#include "hip/hip_runtime.h"
/*
main.cu

This file uses CUDA, cuBLAS and cuDNN to build an LSTM
model for the G-Research Crypto Forecasting dataset.

The cuDNN initialization is follows, roughly, the structure of the 
NVIDIA RNN sample code:
https://developer.nvidia.com/discover/lstm
*/
#include <iostream>
#include <string>
#include <fstream>
#include <vector>
#include <sstream>
#include <hipDNN.h>
#include "hipblas.h"
#include <boost/filesystem.hpp>
#include <boost/algorithm/string.hpp>
using namespace std;


//This will check a hipdnnStatus_t object for failures
#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

/* Kernel to perform MinMax Normalization
	Min-max normalization takes the absolute max/min of the original data and scales to a range
	of the new max/min.
	for a value x[n]:
	x[n]' = (x[n] - min) * (new_max - new_min)
			--------------------------------- + new_min
			 max - min 
*/
__global__ void minmax(float* input, float* output, float newRange, float oldRange, float newMin, float oldMin){
	int idx = (blockIdx.x * blockDim.x) +  threadIdx.x;
	output[idx] = ((((input[idx] - oldMin) * newRange) / oldRange) + newMin);
}

// Method to read in training data from a csv file
// vector<vector<string>>& rows -> pointer to vector to store input data
// string filename -> file name to read (CSV)
// asset -> integer representing asset ID to read
void readCsvFile(vector<vector<string>>& rows, std::string fileName, int asset){
	ifstream filein;
	filein.open(fileName);
	//error if file does not exist
	if (!boost::filesystem::exists(fileName)){
		cerr << "Input file " << fileName << " does not exist" << endl;
		exit(-1);
	}
	//error if not a csv input
	if (!boost::filesystem::extension(fileName).compare(".csv") == 0) {
		cerr << "Input file " << fileName << " must be CSV for training" << endl; 
		exit(-1);
	}
	//error if failed to open file
	if (filein.fail()){
		cerr << "Error opening input file: " << fileName << endl;
		exit(-1);
	}

	vector<string> row;
	string tempRow, value;
	
	// read header
	getline(filein,tempRow);
	
	// read lines from csv file
	while (!filein.eof()){
		getline(filein,tempRow);
		
		// separate row into delimited values
		stringstream str(tempRow);
		row.clear();
		while(getline(str, value, ',')){
			row.push_back(value);
		}
		
		//read data representing the specified asset
		if (stoi(row[1]) == asset){
			
			rows.push_back(row);
		}
	}
	
}

//method to create training sequences and labels
//float* input -> pointer to float array with input data
//vector<float*>& sequences -> pointer to vector of output sequences
//vector<float*>& labels -> pointer to vector of output labels
//int seqSize -> num elements in a sequence
//int labelSize -> num elements in a label
//int inputSize -> num elements in the input data
void createSeqLabels(float* input, vector<float*>& sequences, vector<float*>& labels, int seqSize, int labelSize, int inputSize){
	float* temp;
	float* tempLabel;
	int temp_idx = 0;
	int temp_label= 0;
	cout << "Creating sequences and labels from " << inputSize << " data points" << endl;

	//only segment on data before the train/test split
	for (int index = 0; index < (inputSize - seqSize - labelSize); index+=labelSize){
		temp = new float[seqSize];
		temp_idx = 0;
		tempLabel = new float[labelSize];
		temp_label = 0; 
		
		//create a sequence 
		for (int sub = index; sub < index + seqSize; sub++){
			temp[temp_idx] = input[sub];
			temp_idx++; 
		}

		if (temp_idx == seqSize){
			sequences.push_back(temp);
		}	
	
		//create a label
		for (int sub = index + seqSize; sub < (index + seqSize + labelSize); sub++){
			tempLabel[temp_label] = input[sub];
			temp_label++; 
		}

		if (temp_label == labelSize){
			labels.push_back(tempLabel);
		}	
	}
	
}

//Perform min-max normalization
void normalize(float* inputFeature, float* outputFeature, int splitIdx){
	// setup cublas
	hipblasStatus_t cbstatus;
	hipblasHandle_t cbhandle;
	hipblasCreate(&cbhandle);

	// allocate device memory for input/output
	float* cbInputFeature;
	float* devOutputFeature;
	hipMalloc((void**)&cbInputFeature, splitIdx * sizeof(float));
	hipMalloc((void**)&devOutputFeature, splitIdx * sizeof(float));
		
	//setup min/max variables
	int maxIndex, minIndex;
	float maxVal, minVal;
	float newMax = 1.0;
	float newMin = -1.0;

	//memcopy input to device
	cout << "Copying input feature data to device for CUBLAS" << endl;
	hipMemcpy(cbInputFeature, inputFeature, splitIdx * sizeof(float), hipMemcpyHostToDevice);
	
	//find max val
	cbstatus = hipblasIsamax(cbhandle, splitIdx, cbInputFeature, 1, &maxIndex);
	if( cbstatus != HIPBLAS_STATUS_SUCCESS){
		cerr << "CUBLAS error performing max" << endl;
		return;
	}
	
	//move max value to host
	hipMemcpy(&maxVal, cbInputFeature+maxIndex-1, sizeof(float), hipMemcpyDeviceToHost);
	maxVal = (maxVal >= 0) ? maxVal : -maxVal;
	cout << "Absolute max of input is " << maxVal << endl;
	
	//find min val
	cbstatus = hipblasIsamin(cbhandle, splitIdx, cbInputFeature, 1, &minIndex);
	if( cbstatus != HIPBLAS_STATUS_SUCCESS){
		cerr << "CUBLAS error performing max" << endl;
		return;
	}
	
	//move min value to host
	hipMemcpy(&minVal, cbInputFeature+minIndex-1, sizeof(float), hipMemcpyDeviceToHost);
	minVal = (minVal >= 0) ? minVal : -minVal;
	cout << "Absolute min of input is " << minVal << endl;

	// run normalization kernel
	int blockSize = 256;
	int numBlocks = (splitIdx + blockSize - 1) / blockSize;
	minmax<<<numBlocks, blockSize>>>(cbInputFeature, devOutputFeature, (newMax - newMin), (maxVal - minVal), newMin, minVal);
	
	// copy normalized data to host
	hipMemcpy(outputFeature, devOutputFeature, splitIdx * sizeof(float), hipMemcpyDeviceToHost);
	
	// print sample normalized data
	cout << "Normalization complete. Sample normalization data" << endl << endl;	
	for (int index =0; index < 10; index++){
		cout << "\tOriginal " << inputFeature[index] << " Normalized " << outputFeature[index] << endl;

	}
	cout << endl;
	
	// memory cleanup
	hipFree(cbInputFeature);
	hipFree(&devOutputFeature);
	cbstatus = hipblasDestroy(cbhandle);

	if( cbstatus != HIPBLAS_STATUS_SUCCESS){
		cerr << "CUBLAS shutdown error" << endl;
		return;
	}

}


//main function
int main(int argc, char const *argv[]) {
	int batch = 256; //batch size 
	int miniBatch =256;   	
	int inputSize; //number of input sequences
	int hiddenSize = 100; //hidden cell size
	int numLayers = 1; //number of network layers
	int seqSize = 1440; // number of data points in a sequence
	int labelSize = 60; // number of data points in a label
	//TODO expand code to train on multiple input features
	int features = 1; // number of features to train on
	float splitPercent = 0.8; // train-test data split
	int feature = 3; // index in input data of the feature to train on 
 	int numLinearLayers = 8;
	double paddingFill = 0.0;
 	
	//Ensure the correct number of arguments were passed
	if (argc < 3){
		cerr << "Usage: " << argv[0] << " <mode> <input>" << endl;
		exit(-1);
	}

	//parse train vs eval mode
	string procMode = argv[1];
	boost::to_upper(procMode);
 
	//get input file name
	string inputName = argv[2];	
	int asset = 1;
	
	vector<vector<string>> rows;
	vector<vector<string>> subset;

	//read input data from CSV
	readCsvFile(rows, argv[2], asset);
	cout << "Read " << rows.size() << " input rows" << endl;
	
	cout << "Created subset of data with asset " << asset << " containing " << rows.size() << " data points" << endl;	

	//divide into train/test
	int splitIdx = (int)(splitPercent * rows.size());
	cout << "The train/test split index is " << splitIdx << endl;

	//extract floating point data for the feature
	cout << "Performing Min-Max normaliztation on training data" << endl;
	float* inputFeature = (float*)malloc(splitIdx*sizeof(float));
	float* outputFeature = (float*)malloc(splitIdx*sizeof(float));
	for (int index = 0; index < splitIdx; index++){
		inputFeature[index] = stof(rows[index][feature]);
	}
	
	//perform normalization
	normalize(inputFeature, outputFeature, splitIdx);	

	//create training sequences and labels.
	vector<float*> sequences;
	vector<float*> labels;
	createSeqLabels(outputFeature, sequences, labels, seqSize, labelSize, splitIdx);
	
	cout << "Created " << sequences.size() << " sequences, " << labels.size() << " labels" << endl;

	//set input size to the number of sequences
	inputSize = sequences.size();	

	//setup input sequences on host

	//get device info
	int gpu;
	hipSetDevice(0);
	struct hipDeviceProp_t deviceProperties;
	hipGetDevice(&gpu);
	hipGetDeviceProperties(&deviceProperties, gpu);
	cout << "GPU device properties: " << deviceProperties.name << endl;

	//create CUDNN handle
	hipdnnHandle_t cudnnHandle;
 	checkCUDNN(hipdnnCreate(&cudnnHandle));
	cout << "Created CUDNN handle" << endl;

	//define tensor descriptors
	//tensor desciptors are default initialized to 0
	hipdnnTensorDescriptor_t weightsDescriptor;
	hipdnnTensorDescriptor_t biasDescriptor;
	//hipdnnTensorDescriptor_t inputDescriptor;
    hipdnnTensorDescriptor_t cLongTermDescriptor;
    hipdnnTensorDescriptor_t hShortTermDescriptor;

	//create tensor descriptors
	checkCUDNN(hipdnnCreateTensorDescriptor(&weightsDescriptor));
	checkCUDNN(hipdnnCreateTensorDescriptor(&biasDescriptor));
	//checkCUDNN(hipdnnCreateTensorDescriptor(&inputDescriptor));
	checkCUDNN(hipdnnCreateTensorDescriptor(&cLongTermDescriptor));
    checkCUDNN(hipdnnCreateTensorDescriptor(&hShortTermDescriptor));

	// initialize tensor descriptors
	//params: descriptor, data type, dimensions, size along dimensions, stride along dimensions 
	const int numDimensions = 3; //batch, features, data
    
	//hidden state desc init
	int hiddenDim[numDimensions] = {numLayers, miniBatch, hiddenSize};
	int hiddenStride[3] = {(hiddenDim[1] * hiddenDim[2]), hiddenDim[2], 1};
	checkCUDNN(hipdnnSetTensorNdDescriptor(hShortTermDescriptor, HIPDNN_DATA_FLOAT, numDimensions, hiddenDim, hiddenStride));
    checkCUDNN(hipdnnSetTensorNdDescriptor(cLongTermDescriptor, HIPDNN_DATA_FLOAT, numDimensions, hiddenDim, hiddenStride));
	cout << "Created hidden tensor descriptors" << endl;	
	
	//input desc init
	/*
	int inputDim[numDimensions] = {miniBatch, features, inputSize};
	checkCUDNN(hipdnnSetTensorNdDescriptor(inputDescriptor, CUDA_DATA_FLOAT, numDimensions, inputDim, ));
	cout << "Created input tensor descriptor" << endl;	
	*/

	//define device arrays pointers
	void *deviceInput;
	void *deviceGradInput;
	void *deviceOutput;
	void *deviceGradOutput;
	void *deviceHiddenHX;	
	void *deviceHiddenHY;
	void *deviceHiddenCX;	
	void *deviceHiddenCY;	
	void *deviceHiddenGradHX;	
	void *deviceHiddenGradHY;	
	void *deviceHiddenGradCX;	
	void *deviceHiddenGradCY;	
	int *deviceSequenceLen;

	//allocate device input tensor
	int inputTensorSize = seqSize * miniBatch * inputSize * sizeof(HIPDNN_DATA_FLOAT);
	hipMalloc((void**)&deviceInput, inputTensorSize);
	hipMalloc((void**)&deviceGradInput, inputTensorSize);
	cout << "Allocated input tensors " << endl;

	//allocate device output tensor
	int outputTensorSize = seqSize * miniBatch * hiddenSize * sizeof(HIPDNN_DATA_FLOAT);
	hipMalloc((void**)&deviceOutput, outputTensorSize);
	hipMalloc((void**)&deviceGradOutput, outputTensorSize);
	cout << "Allocated output tensors " << endl;

	//allocate hidden tensors
	int hiddenTensorSize = numLayers * miniBatch * hiddenSize * sizeof(HIPDNN_DATA_FLOAT);

	hipMalloc((void**)&deviceHiddenHX, hiddenTensorSize);
	hipMalloc((void**)&deviceHiddenCX, hiddenTensorSize);
	hipMalloc((void**)&deviceHiddenHY, hiddenTensorSize);
	hipMalloc((void**)&deviceHiddenCY, hiddenTensorSize);
	hipMalloc((void**)&deviceHiddenGradHX, hiddenTensorSize);
	hipMalloc((void**)&deviceHiddenGradCX, hiddenTensorSize);
	hipMalloc((void**)&deviceHiddenGradHY, hiddenTensorSize);
	hipMalloc((void**)&deviceHiddenGradCY, hiddenTensorSize);
	cout << "Allocated output tensors " << endl;

	int* hostSequenceLen = (int*)malloc(miniBatch*sizeof(int));
	for (int i = 0; i < miniBatch; i++){
		hostSequenceLen[i] = seqSize;

	}
	//allocate device input sequence		
	hipMalloc((void **)&deviceSequenceLen, miniBatch * sizeof(int));
	hipMemcpy(deviceSequenceLen, hostSequenceLen, miniBatch * sizeof(int), hipMemcpyHostToDevice);	
	cout << "Allocated input sequence " << endl;

	// Create RNN Data descriptors
	cudnnRNNDataDescriptor_t xRNNDescriptor;
	cudnnRNNDataDescriptor_t yRNNDescriptor;
    checkCUDNN(cudnnCreateRNNDataDescriptor(&xRNNDescriptor));
    checkCUDNN(cudnnCreateRNNDataDescriptor(&yRNNDescriptor));

	// Initialize RNN Data descriptors
    checkCUDNN(cudnnSetRNNDataDescriptor(xRNNDescriptor,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_PACKED,
                                         seqSize,
                                         miniBatch,
                                         inputSize,
                                         hostSequenceLen,
                                         &paddingFill));

    checkCUDNN(cudnnSetRNNDataDescriptor(yRNNDescriptor,
                                         HIPDNN_DATA_FLOAT,
                                         CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_PACKED,
                                         seqSize,
                                         miniBatch,
                                         hiddenSize,
                                         hostSequenceLen,
                                         &paddingFill));

	cout << "Created RNN data descriptors" << endl;

	// Set up the dropout descriptor (needed for the RNN descriptor)
    unsigned long long seed = 1337ull;
	float dropout = 0;
	hipdnnDropoutDescriptor_t dropoutDescriptor;
	size_t stateSize;
    void   *states;
	checkCUDNN(hipdnnCreateDropoutDescriptor(&dropoutDescriptor));
    checkCUDNN(hipdnnDropoutGetStatesSize(cudnnHandle, &stateSize));
   	hipMalloc(&states, stateSize);
    checkCUDNN(hipdnnSetDropoutDescriptor(dropoutDescriptor,
                                            cudnnHandle,
                                            dropout,
                                            states,
                                            stateSize,
                                            seed));




    //Create RNN Descriptor
	hipdnnRNNDescriptor_t RNNDescriptor;
	checkCUDNN(hipdnnCreateRNNDescriptor(&RNNDescriptor));

	//initialize RNN Descriptor
	//no droupout in single layer network
    checkCUDNN(cudnnSetRNNDescriptor_v8(RNNDescriptor,
                                     HIPDNN_RNN_ALGO_PERSIST_DYNAMIC,
                                     HIPDNN_LSTM,
                                     HIPDNN_RNN_NO_BIAS,
                                     HIPDNN_UNIDIRECTIONAL,
                                     HIPDNN_LINEAR_INPUT,
                                     HIPDNN_DATA_FLOAT,
                                     HIPDNN_DATA_FLOAT,
                                     HIPDNN_DEFAULT_MATH,
                                     inputSize,
                                     hiddenSize,
                                     labelSize,
                                     numLayers,
                                     dropoutDescriptor,
                                        0));
	cout << "Created RNN descriptor" << endl;

    // Set up weights and bias parameters
	
	size_t weightSpaceSize;
	void *weightSpace;
	void *gradWeightSpace;
	
    checkCUDNN(cudnnGetRNNWeightSpaceSize(cudnnHandle,RNNDescriptor, &weightSpaceSize));
	hipMalloc((void **)&weightSpace, weightSpaceSize);
    hipMalloc((void **)&gradWeightSpace, weightSpaceSize);
	cout << "Allocated weight space" << endl;

	//Initialize working space and reserved space
	void *workSpace;
    void *reserveSpace;

    size_t workSpaceSize;
    size_t reserveSpaceSize;	

    checkCUDNN(cudnnGetRNNTempSpaceSizes(cudnnHandle,
                                         RNNDescriptor,
                                         CUDNN_FWD_MODE_TRAINING,
                                         xRNNDescriptor,
                                         &workSpaceSize,
                                         &reserveSpaceSize));

    hipMalloc((void **)&workSpace, workSpaceSize);
    hipMalloc((void **)&reserveSpace, reserveSpaceSize);
	hipMemset(gradWeightSpace, 0, weightSpaceSize);	
	
	cout << "Weight space size in MiB: " << weightSpaceSize/1024.0/1024.0 << endl;
	cout << "Work space size in MiB: " << workSpaceSize/1024.0/1024.0 << endl;
	cout << "Reserve space size in MiB: " << reserveSpaceSize/1024.0/1024.0 << endl;

	// Create a dynamic persistent RNN plan
    checkCUDNN(cudnnBuildRNNDynamic(cudnnHandle, RNNDescriptor, batch));
	cout << "Built dynamic persistent RNN plan" << endl;
	
	//Training!
	
	hipDeviceSynchronize();
	//Forward pass
	checkCUDNN(cudnnRNNForward(cudnnHandle,
                               RNNDescriptor,
                               CUDNN_FWD_MODE_TRAINING,
                               deviceSequenceLen,
                               xRNNDescriptor,
                               deviceInput,
                               yRNNDescriptor,
                               deviceOutput,
                               hShortTermDescriptor,
                               deviceHiddenHX,
                               deviceHiddenHY,
                               cLongTermDescriptor,
                               deviceHiddenCX,
                               deviceHiddenCY,
                               weightSpaceSize,
                               weightSpace,
                               workSpaceSize,
                               workSpace,
                               reserveSpaceSize,
                               reserveSpace));

	//Backward pass on data
	checkCUDNN(cudnnRNNBackwardData_v8(cudnnHandle,
                                       RNNDescriptor,
                                       deviceSequenceLen,
                                       yRNNDescriptor,
                                       deviceOutput,
                                       deviceGradOutput,
                                       xRNNDescriptor,
                                       deviceGradInput,
                                       hShortTermDescriptor,
                                       deviceHiddenHX,
                                       deviceHiddenGradHY,
                                       deviceHiddenGradHX,
                                       cLongTermDescriptor,
                                       deviceHiddenCX,
                                       deviceHiddenGradCY,
                                       deviceHiddenGradCX,
                                       weightSpaceSize,
                                       weightSpace,
                                       workSpaceSize,
                                       workSpace,
                                       reserveSpaceSize,
                                       reserveSpace));

	// Backward pass for weights
	checkCUDNN(cudnnRNNBackwardWeights_v8(cudnnHandle,
                                          RNNDescriptor,
                                          CUDNN_WGRAD_MODE_ADD,
                                          deviceSequenceLen,
                                          xRNNDescriptor,
                                          deviceInput,
                                          hShortTermDescriptor,
                                          deviceHiddenHX,
                                          yRNNDescriptor,
                                          deviceOutput,
                                          weightSpaceSize,
                                          gradWeightSpace,
                                          workSpaceSize,
                                          workSpace,
                                          reserveSpaceSize,
                                          reserveSpace));

	hipDeviceSynchronize();
	
	//memory cleanup	
	cout << "Done! cleaning up" << endl;
	free(hostSequenceLen);
	hipFree(deviceSequenceLen);
	hipFree(deviceInput);
	hipFree(deviceGradInput);
	hipFree(deviceOutput);
	hipFree(deviceGradOutput);
	hipFree(deviceHiddenHX);
	hipFree(deviceHiddenHY);
	hipFree(deviceHiddenCX);
	hipFree(deviceHiddenCY);
	hipFree(deviceHiddenGradHX);
	hipFree(deviceHiddenGradHY);
	hipFree(deviceHiddenGradCX);
	hipFree(deviceHiddenGradCY);
	hipFree(workSpace);
	hipFree(reserveSpace);
	hipFree(weightSpace);
	hipFree(gradWeightSpace);
	hipFree(deviceSequenceLen);

	checkCUDNN(cudnnDestroyRNNDataDescriptor(xRNNDescriptor));
	checkCUDNN(cudnnDestroyRNNDataDescriptor(yRNNDescriptor));
	
	checkCUDNN(hipdnnDestroyTensorDescriptor(biasDescriptor));
	checkCUDNN(hipdnnDestroyTensorDescriptor(weightsDescriptor));
	//checkCUDNN(hipdnnDestroyTensorDescriptor(inputDescriptor));
	checkCUDNN(hipdnnDestroyTensorDescriptor(hShortTermDescriptor));
	checkCUDNN(hipdnnDestroyTensorDescriptor(cLongTermDescriptor));
	checkCUDNN(hipdnnDestroyDropoutDescriptor(dropoutDescriptor));	
    checkCUDNN(hipdnnDestroyRNNDescriptor(RNNDescriptor));
    hipdnnDestroy(cudnnHandle);
	
}
